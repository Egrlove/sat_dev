#include "hip/hip_runtime.h"
#define LENX 40
#define LENY 25
#define LENE 25


// #define LENX 8
// #define LENY 5
// #define LENE 5
#define T0 0
#define T1 200
#define T2 50

#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>
#include <fstream>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


static void HandleError(hipError_t err,const char *file,int line)
{
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(error) (HandleError(error, __FILE__, __LINE__))

using namespace std;

void mtrx_print();
void sltn_print();

long int Nm, Nb, Nth, modTime;
long int cntB = 0;
long double *mA, *vX, *vB;
long double L, H, deltaX, deltaT, startT, Tle, Tre;
long double a = 1.0;
int leftEdge, rightEdge;
long int nodePerL, NoX, NoY, edge;

clock_t time_start() { return clock(); }
clock_t time_stop() { return clock(); }

/* Gauss */

void frw_one_th() {
  long int i, j, k;
  long double dgE;

  for (k = 0; k < Nm; k++) {
    dgE = mA[Nm * k + k];
    for (j = k; j < Nm; j++)
      mA[Nm * k + j] /= dgE;
    vB[k] /= dgE;
    // cout << k << endl;
    for (i = k + 1; i < Nm; i++) {
      dgE = mA[Nm * i + k];
      for (j = k; j < Nm; j++)
        mA[Nm * i + j] -= mA[Nm * k + j] * dgE;
      vB[i] -= vB[k] * dgE;
      // cout << vB[i] << endl;
    }
    // cout << endl;
  }
}


__global__ void cuda_gauss_step(long double *dev_mA, long double *dev_vB, int Nm, int k)
{
  double dgE;
  int  j;
  int cuda_i = threadIdx.x + blockIdx.x * blockDim.x;
  if (cuda_i < Nm && cuda_i > k )
  {
    dgE =dev_mA[Nm * (cuda_i) + k] / dev_mA[Nm * (k) + k];
    for (j = k; j < Nm; j++)
      dev_mA[Nm * (cuda_i) + j] -= dev_mA[Nm * k + j] * dgE;
    dev_vB[cuda_i] -= dev_vB[k] * dgE;
  }

}

void cuda_frw_one_th()
{
  long int k;
  // long double dgE;

  long double *dev_mA;
  long double *dev_vB;

  int N_threads = 680;
  int N_blocks;
  if ((Nm % N_threads) == 0)
  {
    N_blocks = (Nm / N_threads);
  }
  else
  {
    N_blocks = (Nm / N_threads) + 1;
  }
  dim3 Threads(N_threads);
  dim3 Blocks(N_blocks);

  HANDLE_ERROR(hipMalloc((void **)&dev_mA, sizeof(long double) * Nm * Nm));
  HANDLE_ERROR(hipMalloc((void **)&dev_vB, sizeof(long double) * Nm));
  
  hipMemcpy(dev_mA, mA, sizeof(long double) * Nm * Nm, hipMemcpyHostToDevice);
  hipMemcpy(dev_vB, vB, sizeof(long double) * Nm, hipMemcpyHostToDevice);
  for (k = 0; k < Nm; k++)
  {
    // dgE = mA[Nm * k + k];
    // for (j = k; j < Nm; j++)
    //   mA[Nm * k + j] /= dgE;
    // vB[k] /= dgE;
    cuda_gauss_step << < Blocks, Threads >> > (dev_mA, dev_vB, Nm, k);
    
  }
  hipMemcpy(mA, dev_mA, sizeof(long double) * Nm* Nm, hipMemcpyDeviceToHost);
  hipMemcpy(vB, dev_vB, sizeof(long double) * Nm, hipMemcpyDeviceToHost);
	HANDLE_ERROR(hipFree(dev_mA));
	HANDLE_ERROR(hipFree(dev_vB));
  
}



void bck_one_th()
{
  long int i, j;

  // vX[Nm - 1] = vB[Nm - 1] / mA[Nm * (Nm - 1) + Nm - 1];
  // for (i = Nm - 2; i >= 0; i--) {
  //   vX[i] = vB[i];
  //   for (j = i + 1; j < Nm; j++)
  //     vX[i] -= mA[Nm * i + j] * vX[j];
  // }

  for(i = Nm - 1; i >= 0; i--){
      long double coeff = mA[i * (Nm) + i];
      vX[i] = vB[i] / coeff;

      for(j = i - 1; j >= 0; j-- ){
          vB[j] -= vB[i] * mA[Nm * j + i] / coeff;
          mA[Nm * j + i] = 0;
      }
  }
}

/* generating */

void all_gen() {
  long int i;
  mA = (long double *)calloc(Nm * Nm, sizeof(long double));
  vX = (long double *)calloc(Nm, sizeof(long double));
  vB = (long double *)calloc(Nm, sizeof(long double));

  for (i = 0; i < Nm; i++) {
    vX[i] = startT;
  }
}

void regen_m_x() {
  long i, j, corr = 0;
  int flag, flagl = 1, crr = 1;

  for (i = 0; i < Nm * Nm; i++)
    mA[i] = 0.0;
  for (i = 0; i < Nm; i++)
    vB[i] = 0.0;
  //
  edge = LENE * nodePerL;
  flag = NoX - edge;
  for (i = 0; i < Nm; i++) {
    vB[i] = vX[i] / deltaT;
    if (i == flagl - 1)
      vB[i] += Tle * a / deltaX * deltaX; // Левая граница
    if (i == flag - 1) {
      vB[i] += Tre * a / deltaX * deltaX; // Правая граница
      flagl = flag + 1;
      flag += NoX - (edge--) + crr;
    }
    if (edge <= 0) {
      edge = 0;
      crr = 0;
    }
  }
  int cnt = 0;
  edge = LENE * nodePerL;
  for (j = 0; j < NoY; j++) {
    for (i = 0; i < NoX - edge; i++) {

      if ((j >= NoY / 4 && j < NoY * 3 / 4 - 1) && (i == NoX / 8)) {
        mA[Nm * (corr + i) + corr + i - 1] = 1;
        mA[Nm * (corr + i) + corr + i] = -(1 + deltaX);
        vB[cnt] = 0.0;
      } else if ((j >= NoY / 4 && j < NoY * 3 / 4 - 1) &&
                 (i == NoX - LENE * nodePerL)) {
        mA[Nm * (corr + i) + corr + i + 1] = 1;
        mA[Nm * (corr + i) + corr + i] = -(1 + deltaX);
        vB[cnt] = 0.0;
      } else {
        mA[Nm * (corr + i) + corr + i] =
            1.0 / deltaT + 2.0 * a / deltaX * deltaX;
        if ((NoX + i) % (NoX) != 0)
          mA[Nm * (corr + i) + corr + i - 1] = -a / deltaX * deltaX;
        if ((NoX - edge + i + 1) % (NoX - edge) != 0)
          mA[Nm * (corr + i) + corr + i + 1] = -a / deltaX * deltaX;
      }

      cnt++;
    }

    corr += NoX - edge;
    if (--edge < 0)
      edge = 0;
  }
}

void regen_m_y() {
  long i, j, corr = 0;
  int cnt = 0, localCnt = 0, cntR = 0;

  for (i = 0; i < Nm * Nm; i++)
    mA[i] = 0.0;
  for (i = 0; i < Nm; i++)
    vB[i] = 0.0;
  for (i = 0; i < Nm; i++) {
    vB[i] = vX[i] / deltaT;
  }
  edge = LENE * nodePerL;
  int oldEdge = edge;
  for (j = 0; j < NoY; j++) {
    for (i = 0; i < NoX - edge; i++) {
      if (((j > 0) && (cnt != NoX - oldEdge + localCnt)) || cntR >= NoX) {
        mA[Nm * (corr + i) + corr + i - (NoX - oldEdge)] = -a / deltaX * deltaX;

        ;
      } else {
        cntR++;
        vB[cnt] += Tre * a / deltaX * deltaX;
      }
      // mA[Nm * (corr + i) + corr + i + (NoX - edge)] = -a / deltaX * deltaX;

      if ((i > NoX / 8 && i < NoX - LENE * nodePerL) &&
          (j >= NoY / 4 && j < NoY * 3 / 4 - 1)) {
        vB[cnt] = 0;
      }
      mA[Nm * (corr + i) + corr + i] = 1.0 / deltaT + 2.0 * a / deltaX * deltaX;

      if (j < NoY - 1) {
        if ((j == NoY / 4 - 1) &&
            (i > NoX / 8 && (i < NoX - LENE * nodePerL))) {
          mA[Nm * (corr + i) + corr + i] = -(1 + deltaX);
          mA[Nm * (corr + i) + corr + i + NoX - edge] = 0;
          mA[Nm * (corr + i) + corr + i - NoX + oldEdge] = 1;
          vB[cnt] = 0;
        } else if ((j == NoY * 3 / 4) &&
                   (i > NoX / 8 && (i < NoX - LENE * nodePerL))) {
          mA[Nm * (corr + i) + corr + i] = 1;
          mA[Nm * (corr + i) + corr + i + NoX - edge] = 0;
          mA[Nm * (corr + i) + corr + i - NoX + oldEdge] = -(1 + deltaX);
          vB[cnt] = 0;
        }
      }

      if (j == NoY - 1) // Теплоизолированность внизу
      {
        vB[cnt] = 0;
        mA[Nm * (corr + i) + corr + NoX - edge + i - (NoX - edge)] = 1;
        if (i == NoX - 1 - edge) {
          mA[Nm * (corr + i) + corr + i - 1 - (NoX - oldEdge)] = -1;
        }
      }

      cnt++;
    }
    corr += NoX - edge;
    localCnt += NoX - edge;
    if (--edge < 0)
      edge = 0;
    if (j > 0)
      oldEdge = edge + 1;
  }
}

/* print */
void mtrx_print() {
  long int i, j;

  for (i = 0; i < Nm; i++) {
    for (j = 0; j < Nm; j++) {
      printf("%.3f ", (double)mA[Nm * i + j]);
      // if (j % NoX == 0 && j != 0)
      //   printf("\n");
    }

    printf("= %.1f", (double)vB[i]);
    printf("\n\n");
  }
}

void sltn_print() {
  long int i;

  for (i = 0; i < Nm; i++) {
    printf("%.3f\t", (double)vX[i]);
  }
  printf("\n");
}

void sltn_pl_print() {
  long int i, j, cnt = 0;

  edge = LENE * nodePerL;
  for (j = 0; j < NoY; j++) {
    for (i = 0; i < NoX - edge; i++) {
      printf("%.1f ", (double)vX[cnt++]);
    }
    printf("\n");
    edge--;
    if (edge < 0)
      edge = 0;
  }
}


/* main */
int main(int argc, char *argv[]) {
  long int i, j, k, ret;
  double cpu_time = 0;
  double gpu_time = 0;

  char string[80];

  ofstream myfile;
  myfile.open("result.txt");

  nodePerL =1;
  modTime = 10;

  Nth = 1; // number of blocks

  L = 1 * LENX;
  H = 1 * LENY;
  Nm = LENY * LENX * nodePerL * nodePerL / 1;
  NoX = LENX * nodePerL;
  NoY = LENY * nodePerL;
  edge = LENE * nodePerL;

  for (i = 1; i <= LENE * nodePerL; i++) {
    Nm -= i;
  }
  // deltaX = H * L / Nm;
  deltaX = 1;

  printf("Step %lf \n", (double)deltaX);

  deltaT = 1;
  startT = T0;
  leftEdge = 1;
  Tle = T1;
  rightEdge = 1;
  Tre = T2;


  printf("Nm: %ld\n", Nm);
  all_gen();

  for (i = 0; i < modTime / deltaT; i++) // 10*
  {
    regen_m_x();
    clock_t start_time = time_start();
    cuda_frw_one_th();
    bck_one_th();
    gpu_time += time_stop() - start_time;

    regen_m_y();

    start_time = time_start();
    cuda_frw_one_th();
    bck_one_th();
    gpu_time += time_stop() - start_time;

    edge = LENE * nodePerL;
    ret = 0;
    for (j = 0; j < NoX - edge + 1; j++) {
      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tre); //Верхняя строка
      // fwrite(string, 1, strlen(string), fds);
      myfile << string ;
    }
    for (j = NoX - edge; j < NoX + 1; j++) {
      // fwrite("-nan ", 1, 5, fds);
      myfile << "-nan ";
    }
    // fwrite("\n", 1, 1, fds);
    myfile << "\n";

    for (j = 0; j < NoY; j++) {
      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tle); // Левая граница
      // fwrite(string, 1, strlen(string), fds);
      myfile << string;

      for (k = 0; k < NoX - edge; k++) {
        string[0] = '\0';
        if ((k > NoX / 8 && k < NoX - LENE * nodePerL) &&
            (j >= NoY / 4 && j < NoY * 3 / 4 - 1)) {
          // fwrite("-nan ", 1, 5, fds);
          myfile << "-nan ";

          ret++;
        } else {
          sprintf(string, "%.3f ", (double)vX[ret++]);
          // fwrite(string, 1, strlen(string), fds);
          myfile << string;
        }
      }

      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tre);
      // fwrite(string, 1, strlen(string), fds);
      myfile << string;

      for (k = NoX - edge; k < NoX; k++) {
        // fwrite("-nan ", 1, 5, fds);
        myfile << "-nan ";
      }
      // fwrite("\n", 1, 1, fds);
      myfile << "\n";

      edge--;
      if (edge < 0)
        edge = 0;
    }
    // fwrite("\n\n", 1, 2, fds);
    myfile << "\n\n";
  }
  printf("GPU solution time: %.3f\n", gpu_time / 1000.0);
  for (i = 0; i < modTime / deltaT; i++) // 10*
  {
    regen_m_x();

    clock_t start_time = time_start();
    frw_one_th();
    bck_one_th();
    cpu_time += time_stop() - start_time;

    regen_m_y();

    start_time = time_start();
    frw_one_th();
    bck_one_th();
    cpu_time += time_stop() - start_time;

    // time += time_stop();

    edge = LENE * nodePerL;
    ret = 0;
    for (j = 0; j < NoX - edge + 1; j++) {
      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tre); //Верхняя строка
      // fwrite(string, 1, strlen(string), fds);
      myfile << string ;
    }
    for (j = NoX - edge; j < NoX + 1; j++) {
      // fwrite("-nan ", 1, 5, fds);
      myfile << "-nan ";
    }
    // fwrite("\n", 1, 1, fds);
    myfile << "\n";

    for (j = 0; j < NoY; j++) {
      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tle); // Левая граница
      // fwrite(string, 1, strlen(string), fds);
      myfile << string;

      for (k = 0; k < NoX - edge; k++) {
        string[0] = '\0';
        if ((k > NoX / 8 && k < NoX - LENE * nodePerL) &&
            (j >= NoY / 4 && j < NoY * 3 / 4 - 1)) {
          // fwrite("-nan ", 1, 5, fds);
          myfile << "-nan ";

          ret++;
        } else {
          sprintf(string, "%.3f ", (double)vX[ret++]);
          // fwrite(string, 1, strlen(string), fds);
          myfile << string;
        }
      }

      string[0] = '\0';
      sprintf(string, "%.3f ", (double)Tre);
      // fwrite(string, 1, strlen(string), fds);
      myfile << string;

      for (k = NoX - edge; k < NoX; k++) {
        // fwrite("-nan ", 1, 5, fds);
        myfile << "-nan ";
      }
      // fwrite("\n", 1, 1, fds);
      myfile << "\n";

      edge--;
      if (edge < 0)
        edge = 0;
    }
    // fwrite("\n\n", 1, 2, fds);
    myfile << "\n\n";
  }
  printf("CPU solution time: %.3f\n", cpu_time / 1000.0);

  printf("TIME RATE: %.3f\n", cpu_time / gpu_time);

  myfile.close();
  free(mA);
  free(vX);
  free(vB);
  return (0);
}
